
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, float* __restrict__ out) {
  float transposed_gemm[2];
  __shared__ float A_shared[32];
  __shared__ float B_shared[128];
  transposed_gemm[0] = 0.000000e+00f;
  transposed_gemm[1] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 128; ++k_outer_outer) {
    __syncthreads();
    A_shared[((int)threadIdx.x)] = A[(((((((int)blockIdx.x) >> 6) * 4096) + ((((int)threadIdx.x) >> 3) * 1024)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    *(float2*)(B_shared + (((int)threadIdx.x) * 2)) = *(float2*)(B + (((((((int)blockIdx.x) & 63) * 16384) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)));
    *(float2*)(B_shared + ((((int)threadIdx.x) * 2) + 64)) = *(float2*)(B + ((((((((int)blockIdx.x) & 63) * 16384) + ((((int)threadIdx.x) >> 2) * 1024)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 8192));
    __syncthreads();
    transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 4) * 16)] * B_shared[((((int)threadIdx.x) & 15) * 8)]));
    transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 1)]));
    transposed_gemm[1] = (transposed_gemm[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 16) + 8)] * B_shared[((((int)threadIdx.x) & 15) * 8)]));
    transposed_gemm[1] = (transposed_gemm[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 1)]));
    transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 2)]));
    transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 3)]));
    transposed_gemm[1] = (transposed_gemm[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 2)]));
    transposed_gemm[1] = (transposed_gemm[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 3)]));
    transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 4)]));
    transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 5)]));
    transposed_gemm[1] = (transposed_gemm[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 4)]));
    transposed_gemm[1] = (transposed_gemm[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 5)]));
    transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 6)]));
    transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 7)]));
    transposed_gemm[1] = (transposed_gemm[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 6)]));
    transposed_gemm[1] = (transposed_gemm[1] + (A_shared[(((((int)threadIdx.x) >> 4) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 7)]));
  }
  for (int i_inner = 0; i_inner < 2; ++i_inner) {
    out[((((((((int)blockIdx.x) >> 6) * 4096) + ((((int)threadIdx.x) >> 4) * 2048)) + (i_inner * 1024)) + ((((int)blockIdx.x) & 63) * 16)) + (((int)threadIdx.x) & 15))] = (transposed_gemm[i_inner] + C[((((((((int)blockIdx.x) >> 6) * 4096) + ((((int)threadIdx.x) >> 4) * 2048)) + (i_inner * 1024)) + ((((int)blockIdx.x) & 63) * 16)) + (((int)threadIdx.x) & 15))]);
  }
}


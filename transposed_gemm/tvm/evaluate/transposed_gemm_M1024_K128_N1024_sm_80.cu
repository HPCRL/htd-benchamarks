
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(64) default_function_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, float* __restrict__ out) {
  float transposed_gemm[32];
  __shared__ float A_shared[128];
  __shared__ float B_shared[1024];
  for (int i_inner_init = 0; i_inner_init < 16; ++i_inner_init) {
    transposed_gemm[(i_inner_init * 2)] = 0.000000e+00f;
    transposed_gemm[((i_inner_init * 2) + 1)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 16; ++k_outer_outer) {
    __syncthreads();
    A_shared[((int)threadIdx.x)] = A[(((((((int)blockIdx.x) >> 3) * 2048) + ((((int)threadIdx.x) >> 3) * 128)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    A_shared[(((int)threadIdx.x) + 64)] = A[((((((((int)blockIdx.x) >> 3) * 2048) + ((((int)threadIdx.x) >> 3) * 128)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7)) + 1024)];
    *(float4*)(B_shared + (((int)threadIdx.x) * 4)) = *(float4*)(B + (((((((int)blockIdx.x) & 7) * 16384) + ((((int)threadIdx.x) >> 1) * 128)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 1) * 4)));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 256)) = *(float4*)(B + ((((((((int)blockIdx.x) & 7) * 16384) + ((((int)threadIdx.x) >> 1) * 128)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 4096));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(B + ((((((((int)blockIdx.x) & 7) * 16384) + ((((int)threadIdx.x) >> 1) * 128)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 8192));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 768)) = *(float4*)(B + ((((((((int)blockIdx.x) & 7) * 16384) + ((((int)threadIdx.x) >> 1) * 128)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 12288));
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int k_inner = 0; k_inner < 4; ++k_inner) {
        for (int i_inner = 0; i_inner < 16; ++i_inner) {
          transposed_gemm[(i_inner * 2)] = (transposed_gemm[(i_inner * 2)] + (A_shared[(((i_inner * 8) + (k_outer_inner * 4)) + k_inner)] * B_shared[(((((int)threadIdx.x) * 16) + (k_outer_inner * 4)) + k_inner)]));
          transposed_gemm[((i_inner * 2) + 1)] = (transposed_gemm[((i_inner * 2) + 1)] + (A_shared[(((i_inner * 8) + (k_outer_inner * 4)) + k_inner)] * B_shared[((((((int)threadIdx.x) * 16) + (k_outer_inner * 4)) + k_inner) + 8)]));
        }
      }
    }
  }
  for (int i_inner_1 = 0; i_inner_1 < 16; ++i_inner_1) {
    for (int j_inner = 0; j_inner < 2; ++j_inner) {
      out[((((((((int)blockIdx.x) >> 3) * 16384) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 7) * 128)) + (((int)threadIdx.x) * 2)) + j_inner)] = (transposed_gemm[((i_inner_1 * 2) + j_inner)] + C[((((((((int)blockIdx.x) >> 3) * 16384) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 7) * 128)) + (((int)threadIdx.x) * 2)) + j_inner)]);
    }
  }
}


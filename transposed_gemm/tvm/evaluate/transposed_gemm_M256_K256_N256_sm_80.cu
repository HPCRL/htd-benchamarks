
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, float* __restrict__ out) {
  float transposed_gemm[8];
  __shared__ float A_shared[256];
  __shared__ float B_shared[256];
  for (int i_outer_inner_init = 0; i_outer_inner_init < 4; ++i_outer_inner_init) {
    for (int i_inner_init = 0; i_inner_init < 2; ++i_inner_init) {
      transposed_gemm[((i_outer_inner_init * 2) + i_inner_init)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 16; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 8; ++ax0_ax1_fused_outer_outer) {
      A_shared[((ax0_ax1_fused_outer_outer * 32) + ((int)threadIdx.x))] = A[((((((((int)blockIdx.x) >> 4) * 4096) + (ax0_ax1_fused_outer_outer * 512)) + ((((int)threadIdx.x) >> 4) * 256)) + (k_outer_outer * 16)) + (((int)threadIdx.x) & 15))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 4; ++ax0_ax1_fused_outer_outer_1) {
      *(float2*)(B_shared + ((ax0_ax1_fused_outer_outer_1 * 64) + (((int)threadIdx.x) * 2))) = *(float2*)(B + ((((((((int)blockIdx.x) & 15) * 4096) + (ax0_ax1_fused_outer_outer_1 * 1024)) + ((((int)threadIdx.x) >> 3) * 256)) + (k_outer_outer * 16)) + ((((int)threadIdx.x) & 7) * 2)));
    }
    __syncthreads();
    for (int i_outer_inner = 0; i_outer_inner < 4; ++i_outer_inner) {
      for (int k_inner = 0; k_inner < 16; ++k_inner) {
        for (int i_inner = 0; i_inner < 2; ++i_inner) {
          transposed_gemm[((i_outer_inner * 2) + i_inner)] = (transposed_gemm[((i_outer_inner * 2) + i_inner)] + (A_shared[(((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 32)) + (i_inner * 16)) + k_inner)] * B_shared[(((((int)threadIdx.x) & 15) * 16) + k_inner)]));
        }
      }
    }
  }
  for (int i_inner_1 = 0; i_inner_1 < 8; ++i_inner_1) {
    out[((((((((int)blockIdx.x) >> 4) * 4096) + ((((int)threadIdx.x) >> 4) * 2048)) + (i_inner_1 * 256)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 15))] = (transposed_gemm[i_inner_1] + C[((((((((int)blockIdx.x) >> 4) * 4096) + ((((int)threadIdx.x) >> 4) * 2048)) + (i_inner_1 * 256)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 15))]);
  }
}


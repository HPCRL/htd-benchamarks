
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) default_function_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, float* __restrict__ out) {
  float transposed_gemm[4];
  __shared__ float A_shared[8192];
  __shared__ float B_shared[4096];
  transposed_gemm[0] = 0.000000e+00f;
  transposed_gemm[1] = 0.000000e+00f;
  transposed_gemm[2] = 0.000000e+00f;
  transposed_gemm[3] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 2; ++k_outer_outer) {
    __syncthreads();
    A_shared[((int)threadIdx.x)] = A[((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x))];
    A_shared[(((int)threadIdx.x) + 128)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 128)];
    A_shared[(((int)threadIdx.x) + 256)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 512)];
    A_shared[(((int)threadIdx.x) + 384)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 640)];
    A_shared[(((int)threadIdx.x) + 512)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 1024)];
    A_shared[(((int)threadIdx.x) + 640)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 1152)];
    A_shared[(((int)threadIdx.x) + 768)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 1536)];
    A_shared[(((int)threadIdx.x) + 896)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 1664)];
    A_shared[(((int)threadIdx.x) + 1024)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 2048)];
    A_shared[(((int)threadIdx.x) + 1152)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 2176)];
    A_shared[(((int)threadIdx.x) + 1280)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 2560)];
    A_shared[(((int)threadIdx.x) + 1408)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 2688)];
    A_shared[(((int)threadIdx.x) + 1536)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 3072)];
    A_shared[(((int)threadIdx.x) + 1664)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 3200)];
    A_shared[(((int)threadIdx.x) + 1792)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 3584)];
    A_shared[(((int)threadIdx.x) + 1920)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 3712)];
    A_shared[(((int)threadIdx.x) + 2048)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 4096)];
    A_shared[(((int)threadIdx.x) + 2176)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 4224)];
    A_shared[(((int)threadIdx.x) + 2304)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 4608)];
    A_shared[(((int)threadIdx.x) + 2432)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 4736)];
    A_shared[(((int)threadIdx.x) + 2560)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 5120)];
    A_shared[(((int)threadIdx.x) + 2688)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 5248)];
    A_shared[(((int)threadIdx.x) + 2816)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 5632)];
    A_shared[(((int)threadIdx.x) + 2944)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 5760)];
    A_shared[(((int)threadIdx.x) + 3072)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 6144)];
    A_shared[(((int)threadIdx.x) + 3200)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 6272)];
    A_shared[(((int)threadIdx.x) + 3328)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 6656)];
    A_shared[(((int)threadIdx.x) + 3456)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 6784)];
    A_shared[(((int)threadIdx.x) + 3584)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 7168)];
    A_shared[(((int)threadIdx.x) + 3712)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 7296)];
    A_shared[(((int)threadIdx.x) + 3840)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 7680)];
    A_shared[(((int)threadIdx.x) + 3968)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 7808)];
    A_shared[(((int)threadIdx.x) + 4096)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 8192)];
    A_shared[(((int)threadIdx.x) + 4224)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 8320)];
    A_shared[(((int)threadIdx.x) + 4352)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 8704)];
    A_shared[(((int)threadIdx.x) + 4480)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 8832)];
    A_shared[(((int)threadIdx.x) + 4608)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 9216)];
    A_shared[(((int)threadIdx.x) + 4736)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 9344)];
    A_shared[(((int)threadIdx.x) + 4864)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 9728)];
    A_shared[(((int)threadIdx.x) + 4992)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 9856)];
    A_shared[(((int)threadIdx.x) + 5120)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 10240)];
    A_shared[(((int)threadIdx.x) + 5248)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 10368)];
    A_shared[(((int)threadIdx.x) + 5376)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 10752)];
    A_shared[(((int)threadIdx.x) + 5504)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 10880)];
    A_shared[(((int)threadIdx.x) + 5632)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 11264)];
    A_shared[(((int)threadIdx.x) + 5760)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 11392)];
    A_shared[(((int)threadIdx.x) + 5888)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 11776)];
    A_shared[(((int)threadIdx.x) + 6016)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 11904)];
    A_shared[(((int)threadIdx.x) + 6144)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 12288)];
    A_shared[(((int)threadIdx.x) + 6272)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 12416)];
    A_shared[(((int)threadIdx.x) + 6400)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 12800)];
    A_shared[(((int)threadIdx.x) + 6528)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 12928)];
    A_shared[(((int)threadIdx.x) + 6656)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 13312)];
    A_shared[(((int)threadIdx.x) + 6784)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 13440)];
    A_shared[(((int)threadIdx.x) + 6912)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 13824)];
    A_shared[(((int)threadIdx.x) + 7040)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 13952)];
    A_shared[(((int)threadIdx.x) + 7168)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 14336)];
    A_shared[(((int)threadIdx.x) + 7296)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 14464)];
    A_shared[(((int)threadIdx.x) + 7424)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 14848)];
    A_shared[(((int)threadIdx.x) + 7552)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 14976)];
    A_shared[(((int)threadIdx.x) + 7680)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 15360)];
    A_shared[(((int)threadIdx.x) + 7808)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 15488)];
    A_shared[(((int)threadIdx.x) + 7936)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 15872)];
    A_shared[(((int)threadIdx.x) + 8064)] = A[(((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 16000)];
    *(float4*)(B_shared + (((int)threadIdx.x) * 4)) = *(float4*)(B + (((((((int)blockIdx.x) % 14) * 8192) + ((((int)threadIdx.x) >> 6) * 512)) + (k_outer_outer * 256)) + ((((int)threadIdx.x) & 63) * 4)));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(B + ((((((((int)blockIdx.x) % 14) * 8192) + ((((int)threadIdx.x) >> 6) * 512)) + (k_outer_outer * 256)) + ((((int)threadIdx.x) & 63) * 4)) + 1024));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 1024)) = *(float4*)(B + ((((((((int)blockIdx.x) % 14) * 8192) + ((((int)threadIdx.x) >> 6) * 512)) + (k_outer_outer * 256)) + ((((int)threadIdx.x) & 63) * 4)) + 2048));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 1536)) = *(float4*)(B + ((((((((int)blockIdx.x) % 14) * 8192) + ((((int)threadIdx.x) >> 6) * 512)) + (k_outer_outer * 256)) + ((((int)threadIdx.x) & 63) * 4)) + 3072));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 2048)) = *(float4*)(B + ((((((((int)blockIdx.x) % 14) * 8192) + ((((int)threadIdx.x) >> 6) * 512)) + (k_outer_outer * 256)) + ((((int)threadIdx.x) & 63) * 4)) + 4096));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 2560)) = *(float4*)(B + ((((((((int)blockIdx.x) % 14) * 8192) + ((((int)threadIdx.x) >> 6) * 512)) + (k_outer_outer * 256)) + ((((int)threadIdx.x) & 63) * 4)) + 5120));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 3072)) = *(float4*)(B + ((((((((int)blockIdx.x) % 14) * 8192) + ((((int)threadIdx.x) >> 6) * 512)) + (k_outer_outer * 256)) + ((((int)threadIdx.x) & 63) * 4)) + 6144));
    *(float4*)(B_shared + ((((int)threadIdx.x) * 4) + 3584)) = *(float4*)(B + ((((((((int)blockIdx.x) % 14) * 8192) + ((((int)threadIdx.x) >> 6) * 512)) + (k_outer_outer * 256)) + ((((int)threadIdx.x) & 63) * 4)) + 7168));
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 256; ++k_outer_inner) {
      transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 1024) + k_outer_inner)] * B_shared[(((((int)threadIdx.x) & 15) * 256) + k_outer_inner)]));
      transposed_gemm[1] = (transposed_gemm[1] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + k_outer_inner) + 256)] * B_shared[(((((int)threadIdx.x) & 15) * 256) + k_outer_inner)]));
      transposed_gemm[2] = (transposed_gemm[2] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + k_outer_inner) + 512)] * B_shared[(((((int)threadIdx.x) & 15) * 256) + k_outer_inner)]));
      transposed_gemm[3] = (transposed_gemm[3] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + k_outer_inner) + 768)] * B_shared[(((((int)threadIdx.x) & 15) * 256) + k_outer_inner)]));
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    out[((((((((int)blockIdx.x) / 14) * 7168) + ((((int)threadIdx.x) >> 4) * 896)) + (i_inner * 224)) + ((((int)blockIdx.x) % 14) * 16)) + (((int)threadIdx.x) & 15))] = (transposed_gemm[i_inner] + C[((((((((int)blockIdx.x) / 14) * 7168) + ((((int)threadIdx.x) >> 4) * 896)) + (i_inner * 224)) + ((((int)blockIdx.x) % 14) * 16)) + (((int)threadIdx.x) & 15))]);
  }
}


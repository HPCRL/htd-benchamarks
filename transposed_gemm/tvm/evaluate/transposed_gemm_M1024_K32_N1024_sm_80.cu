
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(128) default_function_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, float* __restrict__ out) {
  float transposed_gemm[16];
  __shared__ float A_shared[256];
  __shared__ float B_shared[512];
  for (int i_inner_init = 0; i_inner_init < 2; ++i_inner_init) {
    transposed_gemm[i_inner_init] = 0.000000e+00f;
    transposed_gemm[(i_inner_init + 2)] = 0.000000e+00f;
    transposed_gemm[(i_inner_init + 4)] = 0.000000e+00f;
    transposed_gemm[(i_inner_init + 6)] = 0.000000e+00f;
    transposed_gemm[(i_inner_init + 8)] = 0.000000e+00f;
    transposed_gemm[(i_inner_init + 10)] = 0.000000e+00f;
    transposed_gemm[(i_inner_init + 12)] = 0.000000e+00f;
    transposed_gemm[(i_inner_init + 14)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 4; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 2; ++ax0_ax1_fused_outer_outer) {
      A_shared[((ax0_ax1_fused_outer_outer * 128) + ((int)threadIdx.x))] = A[((((((((int)blockIdx.x) >> 4) * 1024) + (ax0_ax1_fused_outer_outer * 512)) + ((((int)threadIdx.x) >> 3) * 32)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 4; ++ax0_ax1_fused_outer_outer_1) {
      B_shared[((ax0_ax1_fused_outer_outer_1 * 128) + ((int)threadIdx.x))] = B[((((((((int)blockIdx.x) & 15) * 2048) + (ax0_ax1_fused_outer_outer_1 * 512)) + ((((int)threadIdx.x) >> 3) * 32)) + (k_outer_outer * 8)) + (((int)threadIdx.x) & 7))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int k_inner = 0; k_inner < 4; ++k_inner) {
        for (int i_inner = 0; i_inner < 2; ++i_inner) {
          transposed_gemm[i_inner] = (transposed_gemm[i_inner] + (A_shared[(((((((int)threadIdx.x) >> 5) * 16) + (i_inner * 8)) + (k_outer_inner * 4)) + k_inner)] * B_shared[((((((int)threadIdx.x) & 31) * 8) + (k_outer_inner * 4)) + k_inner)]));
          transposed_gemm[(i_inner + 2)] = (transposed_gemm[(i_inner + 2)] + (A_shared[(((((((int)threadIdx.x) >> 5) * 16) + (i_inner * 8)) + (k_outer_inner * 4)) + k_inner)] * B_shared[(((((((int)threadIdx.x) & 31) * 8) + (k_outer_inner * 4)) + k_inner) + 256)]));
          transposed_gemm[(i_inner + 4)] = (transposed_gemm[(i_inner + 4)] + (A_shared[((((((((int)threadIdx.x) >> 5) * 16) + (i_inner * 8)) + (k_outer_inner * 4)) + k_inner) + 64)] * B_shared[((((((int)threadIdx.x) & 31) * 8) + (k_outer_inner * 4)) + k_inner)]));
          transposed_gemm[(i_inner + 6)] = (transposed_gemm[(i_inner + 6)] + (A_shared[((((((((int)threadIdx.x) >> 5) * 16) + (i_inner * 8)) + (k_outer_inner * 4)) + k_inner) + 64)] * B_shared[(((((((int)threadIdx.x) & 31) * 8) + (k_outer_inner * 4)) + k_inner) + 256)]));
          transposed_gemm[(i_inner + 8)] = (transposed_gemm[(i_inner + 8)] + (A_shared[((((((((int)threadIdx.x) >> 5) * 16) + (i_inner * 8)) + (k_outer_inner * 4)) + k_inner) + 128)] * B_shared[((((((int)threadIdx.x) & 31) * 8) + (k_outer_inner * 4)) + k_inner)]));
          transposed_gemm[(i_inner + 10)] = (transposed_gemm[(i_inner + 10)] + (A_shared[((((((((int)threadIdx.x) >> 5) * 16) + (i_inner * 8)) + (k_outer_inner * 4)) + k_inner) + 128)] * B_shared[(((((((int)threadIdx.x) & 31) * 8) + (k_outer_inner * 4)) + k_inner) + 256)]));
          transposed_gemm[(i_inner + 12)] = (transposed_gemm[(i_inner + 12)] + (A_shared[((((((((int)threadIdx.x) >> 5) * 16) + (i_inner * 8)) + (k_outer_inner * 4)) + k_inner) + 192)] * B_shared[((((((int)threadIdx.x) & 31) * 8) + (k_outer_inner * 4)) + k_inner)]));
          transposed_gemm[(i_inner + 14)] = (transposed_gemm[(i_inner + 14)] + (A_shared[((((((((int)threadIdx.x) >> 5) * 16) + (i_inner * 8)) + (k_outer_inner * 4)) + k_inner) + 192)] * B_shared[(((((((int)threadIdx.x) & 31) * 8) + (k_outer_inner * 4)) + k_inner) + 256)]));
        }
      }
    }
  }
  for (int i_inner_1 = 0; i_inner_1 < 2; ++i_inner_1) {
    out[((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((int)threadIdx.x) & 31))] = (transposed_gemm[i_inner_1] + C[((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((int)threadIdx.x) & 31))]);
    out[(((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((int)threadIdx.x) & 31)) + 32)] = (transposed_gemm[(i_inner_1 + 2)] + C[(((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((int)threadIdx.x) & 31)) + 32)]);
    out[(((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((int)threadIdx.x) & 31)) + 8192)] = (transposed_gemm[(i_inner_1 + 4)] + C[(((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((int)threadIdx.x) & 31)) + 8192)]);
    out[(((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((int)threadIdx.x) & 31)) + 8224)] = (transposed_gemm[(i_inner_1 + 6)] + C[(((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((int)threadIdx.x) & 31)) + 8224)]);
    out[(((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((int)threadIdx.x) & 31)) + 16384)] = (transposed_gemm[(i_inner_1 + 8)] + C[(((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((int)threadIdx.x) & 31)) + 16384)]);
    out[(((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((int)threadIdx.x) & 31)) + 16416)] = (transposed_gemm[(i_inner_1 + 10)] + C[(((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((int)threadIdx.x) & 31)) + 16416)]);
    out[(((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((int)threadIdx.x) & 31)) + 24576)] = (transposed_gemm[(i_inner_1 + 12)] + C[(((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((int)threadIdx.x) & 31)) + 24576)]);
    out[(((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((int)threadIdx.x) & 31)) + 24608)] = (transposed_gemm[(i_inner_1 + 14)] + C[(((((((((int)blockIdx.x) >> 4) * 32768) + ((((int)threadIdx.x) >> 5) * 2048)) + (i_inner_1 * 1024)) + ((((int)blockIdx.x) & 15) * 64)) + (((int)threadIdx.x) & 31)) + 24608)]);
  }
}


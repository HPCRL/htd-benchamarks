
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(48) default_function_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, float* __restrict__ out) {
  float transposed_gemm[1];
  __shared__ float A_shared[192];
  __shared__ float B_shared[64];
  transposed_gemm[0] = 0.000000e+00f;
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + ((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)));
  B_shared[((int)threadIdx.x)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15))];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1536)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 16));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 16)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1552)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 32));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 32)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1568)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 48));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 48)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1584)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 64));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 64)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1600)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 80));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 80)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1616)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 96));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 96)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1632)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 112));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 112)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1648)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 128));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 128)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1664)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 144));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 144)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1680)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 160));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 160)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1696)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 176));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 176)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1712)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 192));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 192)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1728)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 208));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 208)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1744)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 224));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 224)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1760)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 240));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 240)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1776)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 256));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 256)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1792)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 272));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 272)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1808)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 288));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 288)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1824)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 304));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 304)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1840)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 320));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 320)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1856)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 336));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 336)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1872)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 352));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 352)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1888)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 368));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 368)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1904)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 384));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 384)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1920)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 400));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 400)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1936)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 416));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 416)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1952)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 432));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 432)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1968)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 448));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 448)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 1984)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 464));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 464)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 2000)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 480));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 480)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 2016)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  __syncthreads();
  *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) / 30) * 6144) + ((((int)threadIdx.x) >> 2) * 512)) + ((((int)threadIdx.x) & 3) * 4)) + 496));
  B_shared[((int)threadIdx.x)] = B[(((((((int)blockIdx.x) % 30) * 2048) + ((((int)threadIdx.x) >> 4) * 512)) + (((int)threadIdx.x) & 15)) + 496)];
  if (((int)threadIdx.x) < 16) {
    B_shared[(((int)threadIdx.x) + 48)] = B[((((((int)blockIdx.x) % 30) * 2048) + ((int)threadIdx.x)) + 2032)];
  }
  __syncthreads();
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((int)threadIdx.x) >> 2) * 16)] * B_shared[((((int)threadIdx.x) & 3) * 16)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 1)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 1)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 2)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 2)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 3)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 3)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 4)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 4)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 5)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 5)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 6)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 6)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 7)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 7)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 8)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 8)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 9)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 9)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 10)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 10)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 11)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 11)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 12)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 12)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 13)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 13)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 14)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 14)]));
  transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 2) * 16) + 15)] * B_shared[(((((int)threadIdx.x) & 3) * 16) + 15)]));
  out[(((((((int)blockIdx.x) / 30) * 1440) + ((((int)threadIdx.x) >> 2) * 120)) + ((((int)blockIdx.x) % 30) * 4)) + (((int)threadIdx.x) & 3))] = (transposed_gemm[0] + C[(((((((int)blockIdx.x) / 30) * 1440) + ((((int)threadIdx.x) >> 2) * 120)) + ((((int)blockIdx.x) % 30) * 4)) + (((int)threadIdx.x) & 3))]);
}


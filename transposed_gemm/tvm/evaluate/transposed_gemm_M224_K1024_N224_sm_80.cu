
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, float* __restrict__ out) {
  float transposed_gemm[8];
  __shared__ float A_shared[4096];
  __shared__ float B_shared[4096];
  transposed_gemm[0] = 0.000000e+00f;
  transposed_gemm[4] = 0.000000e+00f;
  transposed_gemm[1] = 0.000000e+00f;
  transposed_gemm[5] = 0.000000e+00f;
  transposed_gemm[2] = 0.000000e+00f;
  transposed_gemm[6] = 0.000000e+00f;
  transposed_gemm[3] = 0.000000e+00f;
  transposed_gemm[7] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 4; ++k_outer_outer) {
    __syncthreads();
    *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + ((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 128)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 128));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 256)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 1024));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 384)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 1152));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 2048));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 640)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 2176));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 768)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 3072));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 896)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 3200));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1024)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 4096));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1152)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 4224));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1280)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 5120));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1408)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 5248));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1536)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 6144));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1664)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 6272));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1792)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 7168));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1920)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 7296));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 2048)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 8192));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 2176)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 8320));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 2304)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 9216));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 2432)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 9344));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 2560)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 10240));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 2688)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 10368));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 2816)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 11264));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 2944)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 11392));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 3072)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 12288));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 3200)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 12416));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 3328)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 13312));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 3456)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 13440));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 3584)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 14336));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 3712)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 14464));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 3840)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 15360));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 3968)) = *(float4*)(A + (((((((int)blockIdx.x) / 14) * 16384) + (k_outer_outer * 256)) + (((int)threadIdx.x) * 4)) + 15488));
    B_shared[((int)threadIdx.x)] = B[((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x))];
    B_shared[(((int)threadIdx.x) + 32)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 32)];
    B_shared[(((int)threadIdx.x) + 64)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 64)];
    B_shared[(((int)threadIdx.x) + 96)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 96)];
    B_shared[(((int)threadIdx.x) + 128)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 128)];
    B_shared[(((int)threadIdx.x) + 160)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 160)];
    B_shared[(((int)threadIdx.x) + 192)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 192)];
    B_shared[(((int)threadIdx.x) + 224)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 224)];
    B_shared[(((int)threadIdx.x) + 256)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 1024)];
    B_shared[(((int)threadIdx.x) + 288)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 1056)];
    B_shared[(((int)threadIdx.x) + 320)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 1088)];
    B_shared[(((int)threadIdx.x) + 352)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 1120)];
    B_shared[(((int)threadIdx.x) + 384)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 1152)];
    B_shared[(((int)threadIdx.x) + 416)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 1184)];
    B_shared[(((int)threadIdx.x) + 448)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 1216)];
    B_shared[(((int)threadIdx.x) + 480)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 1248)];
    B_shared[(((int)threadIdx.x) + 512)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 2048)];
    B_shared[(((int)threadIdx.x) + 544)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 2080)];
    B_shared[(((int)threadIdx.x) + 576)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 2112)];
    B_shared[(((int)threadIdx.x) + 608)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 2144)];
    B_shared[(((int)threadIdx.x) + 640)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 2176)];
    B_shared[(((int)threadIdx.x) + 672)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 2208)];
    B_shared[(((int)threadIdx.x) + 704)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 2240)];
    B_shared[(((int)threadIdx.x) + 736)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 2272)];
    B_shared[(((int)threadIdx.x) + 768)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 3072)];
    B_shared[(((int)threadIdx.x) + 800)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 3104)];
    B_shared[(((int)threadIdx.x) + 832)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 3136)];
    B_shared[(((int)threadIdx.x) + 864)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 3168)];
    B_shared[(((int)threadIdx.x) + 896)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 3200)];
    B_shared[(((int)threadIdx.x) + 928)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 3232)];
    B_shared[(((int)threadIdx.x) + 960)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 3264)];
    B_shared[(((int)threadIdx.x) + 992)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 3296)];
    B_shared[(((int)threadIdx.x) + 1024)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 4096)];
    B_shared[(((int)threadIdx.x) + 1056)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 4128)];
    B_shared[(((int)threadIdx.x) + 1088)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 4160)];
    B_shared[(((int)threadIdx.x) + 1120)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 4192)];
    B_shared[(((int)threadIdx.x) + 1152)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 4224)];
    B_shared[(((int)threadIdx.x) + 1184)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 4256)];
    B_shared[(((int)threadIdx.x) + 1216)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 4288)];
    B_shared[(((int)threadIdx.x) + 1248)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 4320)];
    B_shared[(((int)threadIdx.x) + 1280)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 5120)];
    B_shared[(((int)threadIdx.x) + 1312)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 5152)];
    B_shared[(((int)threadIdx.x) + 1344)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 5184)];
    B_shared[(((int)threadIdx.x) + 1376)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 5216)];
    B_shared[(((int)threadIdx.x) + 1408)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 5248)];
    B_shared[(((int)threadIdx.x) + 1440)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 5280)];
    B_shared[(((int)threadIdx.x) + 1472)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 5312)];
    B_shared[(((int)threadIdx.x) + 1504)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 5344)];
    B_shared[(((int)threadIdx.x) + 1536)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 6144)];
    B_shared[(((int)threadIdx.x) + 1568)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 6176)];
    B_shared[(((int)threadIdx.x) + 1600)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 6208)];
    B_shared[(((int)threadIdx.x) + 1632)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 6240)];
    B_shared[(((int)threadIdx.x) + 1664)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 6272)];
    B_shared[(((int)threadIdx.x) + 1696)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 6304)];
    B_shared[(((int)threadIdx.x) + 1728)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 6336)];
    B_shared[(((int)threadIdx.x) + 1760)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 6368)];
    B_shared[(((int)threadIdx.x) + 1792)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 7168)];
    B_shared[(((int)threadIdx.x) + 1824)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 7200)];
    B_shared[(((int)threadIdx.x) + 1856)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 7232)];
    B_shared[(((int)threadIdx.x) + 1888)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 7264)];
    B_shared[(((int)threadIdx.x) + 1920)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 7296)];
    B_shared[(((int)threadIdx.x) + 1952)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 7328)];
    B_shared[(((int)threadIdx.x) + 1984)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 7360)];
    B_shared[(((int)threadIdx.x) + 2016)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 7392)];
    B_shared[(((int)threadIdx.x) + 2048)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 8192)];
    B_shared[(((int)threadIdx.x) + 2080)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 8224)];
    B_shared[(((int)threadIdx.x) + 2112)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 8256)];
    B_shared[(((int)threadIdx.x) + 2144)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 8288)];
    B_shared[(((int)threadIdx.x) + 2176)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 8320)];
    B_shared[(((int)threadIdx.x) + 2208)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 8352)];
    B_shared[(((int)threadIdx.x) + 2240)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 8384)];
    B_shared[(((int)threadIdx.x) + 2272)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 8416)];
    B_shared[(((int)threadIdx.x) + 2304)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 9216)];
    B_shared[(((int)threadIdx.x) + 2336)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 9248)];
    B_shared[(((int)threadIdx.x) + 2368)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 9280)];
    B_shared[(((int)threadIdx.x) + 2400)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 9312)];
    B_shared[(((int)threadIdx.x) + 2432)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 9344)];
    B_shared[(((int)threadIdx.x) + 2464)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 9376)];
    B_shared[(((int)threadIdx.x) + 2496)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 9408)];
    B_shared[(((int)threadIdx.x) + 2528)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 9440)];
    B_shared[(((int)threadIdx.x) + 2560)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 10240)];
    B_shared[(((int)threadIdx.x) + 2592)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 10272)];
    B_shared[(((int)threadIdx.x) + 2624)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 10304)];
    B_shared[(((int)threadIdx.x) + 2656)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 10336)];
    B_shared[(((int)threadIdx.x) + 2688)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 10368)];
    B_shared[(((int)threadIdx.x) + 2720)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 10400)];
    B_shared[(((int)threadIdx.x) + 2752)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 10432)];
    B_shared[(((int)threadIdx.x) + 2784)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 10464)];
    B_shared[(((int)threadIdx.x) + 2816)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 11264)];
    B_shared[(((int)threadIdx.x) + 2848)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 11296)];
    B_shared[(((int)threadIdx.x) + 2880)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 11328)];
    B_shared[(((int)threadIdx.x) + 2912)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 11360)];
    B_shared[(((int)threadIdx.x) + 2944)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 11392)];
    B_shared[(((int)threadIdx.x) + 2976)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 11424)];
    B_shared[(((int)threadIdx.x) + 3008)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 11456)];
    B_shared[(((int)threadIdx.x) + 3040)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 11488)];
    B_shared[(((int)threadIdx.x) + 3072)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 12288)];
    B_shared[(((int)threadIdx.x) + 3104)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 12320)];
    B_shared[(((int)threadIdx.x) + 3136)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 12352)];
    B_shared[(((int)threadIdx.x) + 3168)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 12384)];
    B_shared[(((int)threadIdx.x) + 3200)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 12416)];
    B_shared[(((int)threadIdx.x) + 3232)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 12448)];
    B_shared[(((int)threadIdx.x) + 3264)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 12480)];
    B_shared[(((int)threadIdx.x) + 3296)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 12512)];
    B_shared[(((int)threadIdx.x) + 3328)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 13312)];
    B_shared[(((int)threadIdx.x) + 3360)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 13344)];
    B_shared[(((int)threadIdx.x) + 3392)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 13376)];
    B_shared[(((int)threadIdx.x) + 3424)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 13408)];
    B_shared[(((int)threadIdx.x) + 3456)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 13440)];
    B_shared[(((int)threadIdx.x) + 3488)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 13472)];
    B_shared[(((int)threadIdx.x) + 3520)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 13504)];
    B_shared[(((int)threadIdx.x) + 3552)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 13536)];
    B_shared[(((int)threadIdx.x) + 3584)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 14336)];
    B_shared[(((int)threadIdx.x) + 3616)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 14368)];
    B_shared[(((int)threadIdx.x) + 3648)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 14400)];
    B_shared[(((int)threadIdx.x) + 3680)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 14432)];
    B_shared[(((int)threadIdx.x) + 3712)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 14464)];
    B_shared[(((int)threadIdx.x) + 3744)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 14496)];
    B_shared[(((int)threadIdx.x) + 3776)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 14528)];
    B_shared[(((int)threadIdx.x) + 3808)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 14560)];
    B_shared[(((int)threadIdx.x) + 3840)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 15360)];
    B_shared[(((int)threadIdx.x) + 3872)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 15392)];
    B_shared[(((int)threadIdx.x) + 3904)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 15424)];
    B_shared[(((int)threadIdx.x) + 3936)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 15456)];
    B_shared[(((int)threadIdx.x) + 3968)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 15488)];
    B_shared[(((int)threadIdx.x) + 4000)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 15520)];
    B_shared[(((int)threadIdx.x) + 4032)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 15552)];
    B_shared[(((int)threadIdx.x) + 4064)] = B[(((((((int)blockIdx.x) % 14) * 16384) + (k_outer_outer * 256)) + ((int)threadIdx.x)) + 15584)];
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 64; ++k_outer_inner) {
      transposed_gemm[0] = (transposed_gemm[0] + (A_shared[(((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4))] * B_shared[(((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4))]));
      transposed_gemm[4] = (transposed_gemm[4] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 2048)] * B_shared[(((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4))]));
      transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 1)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 1)]));
      transposed_gemm[4] = (transposed_gemm[4] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 2049)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 1)]));
      transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 2)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 2)]));
      transposed_gemm[4] = (transposed_gemm[4] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 2050)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 2)]));
      transposed_gemm[0] = (transposed_gemm[0] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 3)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 3)]));
      transposed_gemm[4] = (transposed_gemm[4] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 2051)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 3)]));
      transposed_gemm[1] = (transposed_gemm[1] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 256)] * B_shared[(((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4))]));
      transposed_gemm[5] = (transposed_gemm[5] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 2304)] * B_shared[(((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4))]));
      transposed_gemm[1] = (transposed_gemm[1] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 257)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 1)]));
      transposed_gemm[5] = (transposed_gemm[5] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 2305)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 1)]));
      transposed_gemm[1] = (transposed_gemm[1] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 258)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 2)]));
      transposed_gemm[5] = (transposed_gemm[5] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 2306)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 2)]));
      transposed_gemm[1] = (transposed_gemm[1] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 259)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 3)]));
      transposed_gemm[5] = (transposed_gemm[5] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 2307)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 3)]));
      transposed_gemm[2] = (transposed_gemm[2] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 512)] * B_shared[(((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4))]));
      transposed_gemm[6] = (transposed_gemm[6] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 2560)] * B_shared[(((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4))]));
      transposed_gemm[2] = (transposed_gemm[2] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 513)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 1)]));
      transposed_gemm[6] = (transposed_gemm[6] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 2561)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 1)]));
      transposed_gemm[2] = (transposed_gemm[2] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 514)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 2)]));
      transposed_gemm[6] = (transposed_gemm[6] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 2562)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 2)]));
      transposed_gemm[2] = (transposed_gemm[2] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 515)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 3)]));
      transposed_gemm[6] = (transposed_gemm[6] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 2563)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 3)]));
      transposed_gemm[3] = (transposed_gemm[3] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 768)] * B_shared[(((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4))]));
      transposed_gemm[7] = (transposed_gemm[7] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 2816)] * B_shared[(((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4))]));
      transposed_gemm[3] = (transposed_gemm[3] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 769)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 1)]));
      transposed_gemm[7] = (transposed_gemm[7] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 2817)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 1)]));
      transposed_gemm[3] = (transposed_gemm[3] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 770)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 2)]));
      transposed_gemm[7] = (transposed_gemm[7] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 2818)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 2)]));
      transposed_gemm[3] = (transposed_gemm[3] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 771)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 3)]));
      transposed_gemm[7] = (transposed_gemm[7] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (k_outer_inner * 4)) + 2819)] * B_shared[((((((int)threadIdx.x) & 15) * 256) + (k_outer_inner * 4)) + 3)]));
    }
  }
  for (int i_inner = 0; i_inner < 4; ++i_inner) {
    out[((((((((int)blockIdx.x) / 14) * 3584) + ((((int)threadIdx.x) >> 4) * 896)) + (i_inner * 224)) + ((((int)blockIdx.x) % 14) * 16)) + (((int)threadIdx.x) & 15))] = (transposed_gemm[i_inner] + C[((((((((int)blockIdx.x) / 14) * 3584) + ((((int)threadIdx.x) >> 4) * 896)) + (i_inner * 224)) + ((((int)blockIdx.x) % 14) * 16)) + (((int)threadIdx.x) & 15))]);
    out[(((((((((int)blockIdx.x) / 14) * 3584) + ((((int)threadIdx.x) >> 4) * 896)) + (i_inner * 224)) + ((((int)blockIdx.x) % 14) * 16)) + (((int)threadIdx.x) & 15)) + 1792)] = (transposed_gemm[(i_inner + 4)] + C[(((((((((int)blockIdx.x) / 14) * 3584) + ((((int)threadIdx.x) >> 4) * 896)) + (i_inner * 224)) + ((((int)blockIdx.x) % 14) * 16)) + (((int)threadIdx.x) & 15)) + 1792)]);
  }
}


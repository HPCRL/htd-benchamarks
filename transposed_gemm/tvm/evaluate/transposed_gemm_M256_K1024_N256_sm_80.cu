
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, float* __restrict__ out) {
  float transposed_gemm[8];
  __shared__ float A_shared[2048];
  __shared__ float B_shared[2048];
  transposed_gemm[0] = 0.000000e+00f;
  transposed_gemm[1] = 0.000000e+00f;
  transposed_gemm[2] = 0.000000e+00f;
  transposed_gemm[3] = 0.000000e+00f;
  transposed_gemm[4] = 0.000000e+00f;
  transposed_gemm[5] = 0.000000e+00f;
  transposed_gemm[6] = 0.000000e+00f;
  transposed_gemm[7] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 8; ++k_outer_outer) {
    __syncthreads();
    *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + ((((((int)blockIdx.x) >> 4) * 16384) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 128)) = *(float4*)(A + (((((((int)blockIdx.x) >> 4) * 16384) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 1024));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 256)) = *(float4*)(A + (((((((int)blockIdx.x) >> 4) * 16384) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 2048));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 384)) = *(float4*)(A + (((((((int)blockIdx.x) >> 4) * 16384) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 3072));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 512)) = *(float4*)(A + (((((((int)blockIdx.x) >> 4) * 16384) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 4096));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 640)) = *(float4*)(A + (((((((int)blockIdx.x) >> 4) * 16384) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 5120));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 768)) = *(float4*)(A + (((((((int)blockIdx.x) >> 4) * 16384) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 6144));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 896)) = *(float4*)(A + (((((((int)blockIdx.x) >> 4) * 16384) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 7168));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1024)) = *(float4*)(A + (((((((int)blockIdx.x) >> 4) * 16384) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 8192));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1152)) = *(float4*)(A + (((((((int)blockIdx.x) >> 4) * 16384) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 9216));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1280)) = *(float4*)(A + (((((((int)blockIdx.x) >> 4) * 16384) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 10240));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1408)) = *(float4*)(A + (((((((int)blockIdx.x) >> 4) * 16384) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 11264));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1536)) = *(float4*)(A + (((((((int)blockIdx.x) >> 4) * 16384) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 12288));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1664)) = *(float4*)(A + (((((((int)blockIdx.x) >> 4) * 16384) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 13312));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1792)) = *(float4*)(A + (((((((int)blockIdx.x) >> 4) * 16384) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 14336));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 1920)) = *(float4*)(A + (((((((int)blockIdx.x) >> 4) * 16384) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)) + 15360));
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 32; ++ax0_ax1_fused_outer_outer) {
      *(float2*)(B_shared + ((ax0_ax1_fused_outer_outer * 64) + (((int)threadIdx.x) * 2))) = *(float2*)(B + ((((((((int)blockIdx.x) & 15) * 16384) + ((ax0_ax1_fused_outer_outer >> 1) * 1024)) + (k_outer_outer * 128)) + ((ax0_ax1_fused_outer_outer & 1) * 64)) + (((int)threadIdx.x) * 2)));
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 16; ++k_outer_inner) {
      for (int i_outer_inner = 0; i_outer_inner < 8; ++i_outer_inner) {
        transposed_gemm[i_outer_inner] = (transposed_gemm[i_outer_inner] + (A_shared[((((((int)threadIdx.x) >> 4) * 1024) + (i_outer_inner * 128)) + (k_outer_inner * 8))] * B_shared[(((((int)threadIdx.x) & 15) * 128) + (k_outer_inner * 8))]));
        transposed_gemm[i_outer_inner] = (transposed_gemm[i_outer_inner] + (A_shared[(((((((int)threadIdx.x) >> 4) * 1024) + (i_outer_inner * 128)) + (k_outer_inner * 8)) + 1)] * B_shared[((((((int)threadIdx.x) & 15) * 128) + (k_outer_inner * 8)) + 1)]));
        transposed_gemm[i_outer_inner] = (transposed_gemm[i_outer_inner] + (A_shared[(((((((int)threadIdx.x) >> 4) * 1024) + (i_outer_inner * 128)) + (k_outer_inner * 8)) + 2)] * B_shared[((((((int)threadIdx.x) & 15) * 128) + (k_outer_inner * 8)) + 2)]));
        transposed_gemm[i_outer_inner] = (transposed_gemm[i_outer_inner] + (A_shared[(((((((int)threadIdx.x) >> 4) * 1024) + (i_outer_inner * 128)) + (k_outer_inner * 8)) + 3)] * B_shared[((((((int)threadIdx.x) & 15) * 128) + (k_outer_inner * 8)) + 3)]));
        transposed_gemm[i_outer_inner] = (transposed_gemm[i_outer_inner] + (A_shared[(((((((int)threadIdx.x) >> 4) * 1024) + (i_outer_inner * 128)) + (k_outer_inner * 8)) + 4)] * B_shared[((((((int)threadIdx.x) & 15) * 128) + (k_outer_inner * 8)) + 4)]));
        transposed_gemm[i_outer_inner] = (transposed_gemm[i_outer_inner] + (A_shared[(((((((int)threadIdx.x) >> 4) * 1024) + (i_outer_inner * 128)) + (k_outer_inner * 8)) + 5)] * B_shared[((((((int)threadIdx.x) & 15) * 128) + (k_outer_inner * 8)) + 5)]));
        transposed_gemm[i_outer_inner] = (transposed_gemm[i_outer_inner] + (A_shared[(((((((int)threadIdx.x) >> 4) * 1024) + (i_outer_inner * 128)) + (k_outer_inner * 8)) + 6)] * B_shared[((((((int)threadIdx.x) & 15) * 128) + (k_outer_inner * 8)) + 6)]));
        transposed_gemm[i_outer_inner] = (transposed_gemm[i_outer_inner] + (A_shared[(((((((int)threadIdx.x) >> 4) * 1024) + (i_outer_inner * 128)) + (k_outer_inner * 8)) + 7)] * B_shared[((((((int)threadIdx.x) & 15) * 128) + (k_outer_inner * 8)) + 7)]));
      }
    }
  }
  for (int i_inner = 0; i_inner < 8; ++i_inner) {
    out[((((((((int)blockIdx.x) >> 4) * 4096) + ((((int)threadIdx.x) >> 4) * 2048)) + (i_inner * 256)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 15))] = (transposed_gemm[i_inner] + C[((((((((int)blockIdx.x) >> 4) * 4096) + ((((int)threadIdx.x) >> 4) * 2048)) + (i_inner * 256)) + ((((int)blockIdx.x) & 15) * 16)) + (((int)threadIdx.x) & 15))]);
  }
}


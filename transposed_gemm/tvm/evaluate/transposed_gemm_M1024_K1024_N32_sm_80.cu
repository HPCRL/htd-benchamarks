
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, float* __restrict__ out) {
  float transposed_gemm[8];
  __shared__ float A_shared[2048];
  __shared__ float B_shared[2048];
  for (int j_outer_inner_init = 0; j_outer_inner_init < 4; ++j_outer_inner_init) {
    transposed_gemm[j_outer_inner_init] = 0.000000e+00f;
    transposed_gemm[(j_outer_inner_init + 4)] = 0.000000e+00f;
  }
  for (int k_outer_outer = 0; k_outer_outer < 8; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 16; ++ax0_ax1_fused_outer_outer) {
      *(float4*)(A_shared + ((ax0_ax1_fused_outer_outer * 128) + (((int)threadIdx.x) * 4))) = *(float4*)(A + (((((((int)blockIdx.x) >> 1) * 16384) + (ax0_ax1_fused_outer_outer * 1024)) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)));
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 16; ++ax0_ax1_fused_outer_outer_1) {
      *(float4*)(B_shared + ((ax0_ax1_fused_outer_outer_1 * 128) + (((int)threadIdx.x) * 4))) = *(float4*)(B + (((((((int)blockIdx.x) & 1) * 16384) + (ax0_ax1_fused_outer_outer_1 * 1024)) + (k_outer_outer * 128)) + (((int)threadIdx.x) * 4)));
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 16; ++k_outer_inner) {
      for (int j_outer_inner = 0; j_outer_inner < 4; ++j_outer_inner) {
        for (int k_inner = 0; k_inner < 8; ++k_inner) {
          transposed_gemm[j_outer_inner] = (transposed_gemm[j_outer_inner] + (A_shared[((((((int)threadIdx.x) >> 1) * 128) + (k_outer_inner * 8)) + k_inner)] * B_shared[(((((((int)threadIdx.x) & 1) * 512) + (j_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner)]));
          transposed_gemm[(j_outer_inner + 4)] = (transposed_gemm[(j_outer_inner + 4)] + (A_shared[((((((int)threadIdx.x) >> 1) * 128) + (k_outer_inner * 8)) + k_inner)] * B_shared[((((((((int)threadIdx.x) & 1) * 512) + (j_outer_inner * 128)) + (k_outer_inner * 8)) + k_inner) + 1024)]));
        }
      }
    }
  }
  for (int j_inner = 0; j_inner < 4; ++j_inner) {
    out[((((((((int)blockIdx.x) >> 1) * 512) + ((((int)threadIdx.x) >> 1) * 32)) + ((((int)blockIdx.x) & 1) * 16)) + ((((int)threadIdx.x) & 1) * 4)) + j_inner)] = (transposed_gemm[j_inner] + C[((((((((int)blockIdx.x) >> 1) * 512) + ((((int)threadIdx.x) >> 1) * 32)) + ((((int)blockIdx.x) & 1) * 16)) + ((((int)threadIdx.x) & 1) * 4)) + j_inner)]);
    out[(((((((((int)blockIdx.x) >> 1) * 512) + ((((int)threadIdx.x) >> 1) * 32)) + ((((int)blockIdx.x) & 1) * 16)) + ((((int)threadIdx.x) & 1) * 4)) + j_inner) + 8)] = (transposed_gemm[(j_inner + 4)] + C[(((((((((int)blockIdx.x) >> 1) * 512) + ((((int)threadIdx.x) >> 1) * 32)) + ((((int)blockIdx.x) & 1) * 16)) + ((((int)threadIdx.x) & 1) * 4)) + j_inner) + 8)]);
  }
}


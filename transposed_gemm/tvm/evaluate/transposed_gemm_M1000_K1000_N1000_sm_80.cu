
#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(250) default_function_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, float* __restrict__ out) {
  float transposed_gemm[8];
  __shared__ float A_shared[4000];
  __shared__ float B_shared[5000];
  for (int i_outer_inner_init = 0; i_outer_inner_init < 2; ++i_outer_inner_init) {
    for (int i_inner_init = 0; i_inner_init < 2; ++i_inner_init) {
      transposed_gemm[((i_outer_inner_init * 2) + i_inner_init)] = 0.000000e+00f;
      transposed_gemm[(((i_outer_inner_init * 2) + i_inner_init) + 4)] = 0.000000e+00f;
    }
  }
  for (int k_outer_outer = 0; k_outer_outer < 10; ++k_outer_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_outer_outer = 0; ax0_ax1_fused_outer_outer < 8; ++ax0_ax1_fused_outer_outer) {
      *(float2*)(A_shared + ((ax0_ax1_fused_outer_outer * 500) + (((int)threadIdx.x) * 2))) = *(float2*)(A + ((((((((int)blockIdx.x) / 20) * 40000) + (ax0_ax1_fused_outer_outer * 5000)) + ((((int)threadIdx.x) / 50) * 1000)) + (k_outer_outer * 100)) + ((((int)threadIdx.x) % 50) * 2)));
    }
    for (int ax0_ax1_fused_outer_outer_1 = 0; ax0_ax1_fused_outer_outer_1 < 20; ++ax0_ax1_fused_outer_outer_1) {
      B_shared[((ax0_ax1_fused_outer_outer_1 * 250) + ((int)threadIdx.x))] = B[(((((((int)blockIdx.x) % 20) * 50000) + ((((ax0_ax1_fused_outer_outer_1 * 5) + (((int)threadIdx.x) / 50)) >> 1) * 1000)) + (k_outer_outer * 100)) + (((ax0_ax1_fused_outer_outer_1 * 50) + ((int)threadIdx.x)) % 100))];
    }
    __syncthreads();
    for (int k_outer_inner = 0; k_outer_inner < 2; ++k_outer_inner) {
      for (int i_outer_inner = 0; i_outer_inner < 2; ++i_outer_inner) {
        for (int k_inner = 0; k_inner < 50; ++k_inner) {
          for (int i_inner = 0; i_inner < 2; ++i_inner) {
            transposed_gemm[((i_outer_inner * 2) + i_inner)] = (transposed_gemm[((i_outer_inner * 2) + i_inner)] + (A_shared[((((((((int)threadIdx.x) / 50) * 400) + (i_outer_inner * 200)) + (i_inner * 100)) + (k_outer_inner * 50)) + k_inner)] * B_shared[((((((int)threadIdx.x) % 50) * 100) + (k_outer_inner * 50)) + k_inner)]));
            transposed_gemm[(((i_outer_inner * 2) + i_inner) + 4)] = (transposed_gemm[(((i_outer_inner * 2) + i_inner) + 4)] + (A_shared[(((((((((int)threadIdx.x) / 50) * 400) + (i_outer_inner * 200)) + (i_inner * 100)) + (k_outer_inner * 50)) + k_inner) + 2000)] * B_shared[((((((int)threadIdx.x) % 50) * 100) + (k_outer_inner * 50)) + k_inner)]));
          }
        }
      }
    }
  }
  for (int i_inner_1 = 0; i_inner_1 < 4; ++i_inner_1) {
    out[((((((((int)blockIdx.x) / 20) * 40000) + ((((int)threadIdx.x) / 50) * 4000)) + (i_inner_1 * 1000)) + ((((int)blockIdx.x) % 20) * 50)) + (((int)threadIdx.x) % 50))] = (transposed_gemm[i_inner_1] + C[((((((((int)blockIdx.x) / 20) * 40000) + ((((int)threadIdx.x) / 50) * 4000)) + (i_inner_1 * 1000)) + ((((int)blockIdx.x) % 20) * 50)) + (((int)threadIdx.x) % 50))]);
    out[(((((((((int)blockIdx.x) / 20) * 40000) + ((((int)threadIdx.x) / 50) * 4000)) + (i_inner_1 * 1000)) + ((((int)blockIdx.x) % 20) * 50)) + (((int)threadIdx.x) % 50)) + 20000)] = (transposed_gemm[(i_inner_1 + 4)] + C[(((((((((int)blockIdx.x) / 20) * 40000) + ((((int)threadIdx.x) / 50) * 4000)) + (i_inner_1 * 1000)) + ((((int)blockIdx.x) % 20) * 50)) + (((int)threadIdx.x) % 50)) + 20000)]);
  }
}



#include <hip/hip_runtime.h>

#if (((__CUDACC_VER_MAJOR__ == 11) && (__CUDACC_VER_MINOR__ >= 4)) || \
     (__CUDACC_VER_MAJOR__ > 11))
#define TVM_ENABLE_L2_PREFETCH 1
#else
#define TVM_ENABLE_L2_PREFETCH 0
#endif

#ifdef _WIN32
  using uint = unsigned int;
  using uchar = unsigned char;
  using ushort = unsigned short;
  using int64_t = long long;
  using uint64_t = unsigned long long;
#else
  #define uint unsigned int
  #define uchar unsigned char
  #define ushort unsigned short
  #define int64_t long long
  #define uint64_t unsigned long long
#endif
extern "C" __global__ void __launch_bounds__(32) default_function_kernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, float* __restrict__ out) {
  float transposed_gemm[16];
  __shared__ float A_shared[256];
  __shared__ float B_shared[128];
  transposed_gemm[0] = 0.000000e+00f;
  transposed_gemm[1] = 0.000000e+00f;
  transposed_gemm[2] = 0.000000e+00f;
  transposed_gemm[3] = 0.000000e+00f;
  transposed_gemm[4] = 0.000000e+00f;
  transposed_gemm[5] = 0.000000e+00f;
  transposed_gemm[6] = 0.000000e+00f;
  transposed_gemm[7] = 0.000000e+00f;
  transposed_gemm[8] = 0.000000e+00f;
  transposed_gemm[9] = 0.000000e+00f;
  transposed_gemm[10] = 0.000000e+00f;
  transposed_gemm[11] = 0.000000e+00f;
  transposed_gemm[12] = 0.000000e+00f;
  transposed_gemm[13] = 0.000000e+00f;
  transposed_gemm[14] = 0.000000e+00f;
  transposed_gemm[15] = 0.000000e+00f;
  for (int k_outer_outer = 0; k_outer_outer < 64; ++k_outer_outer) {
    __syncthreads();
    *(float4*)(A_shared + (((int)threadIdx.x) * 4)) = *(float4*)(A + (((((((int)blockIdx.x) >> 6) * 16384) + ((((int)threadIdx.x) >> 1) * 512)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 1) * 4)));
    *(float4*)(A_shared + ((((int)threadIdx.x) * 4) + 128)) = *(float4*)(A + ((((((((int)blockIdx.x) >> 6) * 16384) + ((((int)threadIdx.x) >> 1) * 512)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 1) * 4)) + 8192));
    *(float2*)(B_shared + (((int)threadIdx.x) * 2)) = *(float2*)(B + (((((((int)blockIdx.x) & 63) * 8192) + ((((int)threadIdx.x) >> 2) * 512)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)));
    *(float2*)(B_shared + ((((int)threadIdx.x) * 2) + 64)) = *(float2*)(B + ((((((((int)blockIdx.x) & 63) * 8192) + ((((int)threadIdx.x) >> 2) * 512)) + (k_outer_outer * 8)) + ((((int)threadIdx.x) & 3) * 2)) + 4096));
    __syncthreads();
    for (int i_outer_inner = 0; i_outer_inner < 2; ++i_outer_inner) {
      transposed_gemm[(i_outer_inner * 8)] = (transposed_gemm[(i_outer_inner * 8)] + (A_shared[(((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64))] * B_shared[((((int)threadIdx.x) & 15) * 8)]));
      transposed_gemm[((i_outer_inner * 8) + 1)] = (transposed_gemm[((i_outer_inner * 8) + 1)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 8)] * B_shared[((((int)threadIdx.x) & 15) * 8)]));
      transposed_gemm[((i_outer_inner * 8) + 2)] = (transposed_gemm[((i_outer_inner * 8) + 2)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 16)] * B_shared[((((int)threadIdx.x) & 15) * 8)]));
      transposed_gemm[((i_outer_inner * 8) + 3)] = (transposed_gemm[((i_outer_inner * 8) + 3)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 24)] * B_shared[((((int)threadIdx.x) & 15) * 8)]));
      transposed_gemm[((i_outer_inner * 8) + 4)] = (transposed_gemm[((i_outer_inner * 8) + 4)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 32)] * B_shared[((((int)threadIdx.x) & 15) * 8)]));
      transposed_gemm[((i_outer_inner * 8) + 5)] = (transposed_gemm[((i_outer_inner * 8) + 5)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 40)] * B_shared[((((int)threadIdx.x) & 15) * 8)]));
      transposed_gemm[((i_outer_inner * 8) + 6)] = (transposed_gemm[((i_outer_inner * 8) + 6)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 48)] * B_shared[((((int)threadIdx.x) & 15) * 8)]));
      transposed_gemm[((i_outer_inner * 8) + 7)] = (transposed_gemm[((i_outer_inner * 8) + 7)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 56)] * B_shared[((((int)threadIdx.x) & 15) * 8)]));
      transposed_gemm[(i_outer_inner * 8)] = (transposed_gemm[(i_outer_inner * 8)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 1)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 1)]));
      transposed_gemm[((i_outer_inner * 8) + 1)] = (transposed_gemm[((i_outer_inner * 8) + 1)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 9)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 1)]));
      transposed_gemm[((i_outer_inner * 8) + 2)] = (transposed_gemm[((i_outer_inner * 8) + 2)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 17)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 1)]));
      transposed_gemm[((i_outer_inner * 8) + 3)] = (transposed_gemm[((i_outer_inner * 8) + 3)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 25)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 1)]));
      transposed_gemm[((i_outer_inner * 8) + 4)] = (transposed_gemm[((i_outer_inner * 8) + 4)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 33)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 1)]));
      transposed_gemm[((i_outer_inner * 8) + 5)] = (transposed_gemm[((i_outer_inner * 8) + 5)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 41)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 1)]));
      transposed_gemm[((i_outer_inner * 8) + 6)] = (transposed_gemm[((i_outer_inner * 8) + 6)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 49)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 1)]));
      transposed_gemm[((i_outer_inner * 8) + 7)] = (transposed_gemm[((i_outer_inner * 8) + 7)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 57)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 1)]));
      transposed_gemm[(i_outer_inner * 8)] = (transposed_gemm[(i_outer_inner * 8)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 2)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 2)]));
      transposed_gemm[((i_outer_inner * 8) + 1)] = (transposed_gemm[((i_outer_inner * 8) + 1)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 10)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 2)]));
      transposed_gemm[((i_outer_inner * 8) + 2)] = (transposed_gemm[((i_outer_inner * 8) + 2)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 18)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 2)]));
      transposed_gemm[((i_outer_inner * 8) + 3)] = (transposed_gemm[((i_outer_inner * 8) + 3)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 26)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 2)]));
      transposed_gemm[((i_outer_inner * 8) + 4)] = (transposed_gemm[((i_outer_inner * 8) + 4)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 34)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 2)]));
      transposed_gemm[((i_outer_inner * 8) + 5)] = (transposed_gemm[((i_outer_inner * 8) + 5)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 42)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 2)]));
      transposed_gemm[((i_outer_inner * 8) + 6)] = (transposed_gemm[((i_outer_inner * 8) + 6)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 50)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 2)]));
      transposed_gemm[((i_outer_inner * 8) + 7)] = (transposed_gemm[((i_outer_inner * 8) + 7)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 58)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 2)]));
      transposed_gemm[(i_outer_inner * 8)] = (transposed_gemm[(i_outer_inner * 8)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 3)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 3)]));
      transposed_gemm[((i_outer_inner * 8) + 1)] = (transposed_gemm[((i_outer_inner * 8) + 1)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 11)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 3)]));
      transposed_gemm[((i_outer_inner * 8) + 2)] = (transposed_gemm[((i_outer_inner * 8) + 2)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 19)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 3)]));
      transposed_gemm[((i_outer_inner * 8) + 3)] = (transposed_gemm[((i_outer_inner * 8) + 3)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 27)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 3)]));
      transposed_gemm[((i_outer_inner * 8) + 4)] = (transposed_gemm[((i_outer_inner * 8) + 4)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 35)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 3)]));
      transposed_gemm[((i_outer_inner * 8) + 5)] = (transposed_gemm[((i_outer_inner * 8) + 5)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 43)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 3)]));
      transposed_gemm[((i_outer_inner * 8) + 6)] = (transposed_gemm[((i_outer_inner * 8) + 6)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 51)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 3)]));
      transposed_gemm[((i_outer_inner * 8) + 7)] = (transposed_gemm[((i_outer_inner * 8) + 7)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 59)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 3)]));
      transposed_gemm[(i_outer_inner * 8)] = (transposed_gemm[(i_outer_inner * 8)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 4)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 4)]));
      transposed_gemm[((i_outer_inner * 8) + 1)] = (transposed_gemm[((i_outer_inner * 8) + 1)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 12)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 4)]));
      transposed_gemm[((i_outer_inner * 8) + 2)] = (transposed_gemm[((i_outer_inner * 8) + 2)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 20)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 4)]));
      transposed_gemm[((i_outer_inner * 8) + 3)] = (transposed_gemm[((i_outer_inner * 8) + 3)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 28)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 4)]));
      transposed_gemm[((i_outer_inner * 8) + 4)] = (transposed_gemm[((i_outer_inner * 8) + 4)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 36)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 4)]));
      transposed_gemm[((i_outer_inner * 8) + 5)] = (transposed_gemm[((i_outer_inner * 8) + 5)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 44)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 4)]));
      transposed_gemm[((i_outer_inner * 8) + 6)] = (transposed_gemm[((i_outer_inner * 8) + 6)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 52)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 4)]));
      transposed_gemm[((i_outer_inner * 8) + 7)] = (transposed_gemm[((i_outer_inner * 8) + 7)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 60)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 4)]));
      transposed_gemm[(i_outer_inner * 8)] = (transposed_gemm[(i_outer_inner * 8)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 5)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 5)]));
      transposed_gemm[((i_outer_inner * 8) + 1)] = (transposed_gemm[((i_outer_inner * 8) + 1)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 13)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 5)]));
      transposed_gemm[((i_outer_inner * 8) + 2)] = (transposed_gemm[((i_outer_inner * 8) + 2)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 21)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 5)]));
      transposed_gemm[((i_outer_inner * 8) + 3)] = (transposed_gemm[((i_outer_inner * 8) + 3)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 29)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 5)]));
      transposed_gemm[((i_outer_inner * 8) + 4)] = (transposed_gemm[((i_outer_inner * 8) + 4)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 37)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 5)]));
      transposed_gemm[((i_outer_inner * 8) + 5)] = (transposed_gemm[((i_outer_inner * 8) + 5)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 45)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 5)]));
      transposed_gemm[((i_outer_inner * 8) + 6)] = (transposed_gemm[((i_outer_inner * 8) + 6)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 53)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 5)]));
      transposed_gemm[((i_outer_inner * 8) + 7)] = (transposed_gemm[((i_outer_inner * 8) + 7)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 61)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 5)]));
      transposed_gemm[(i_outer_inner * 8)] = (transposed_gemm[(i_outer_inner * 8)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 6)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 6)]));
      transposed_gemm[((i_outer_inner * 8) + 1)] = (transposed_gemm[((i_outer_inner * 8) + 1)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 14)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 6)]));
      transposed_gemm[((i_outer_inner * 8) + 2)] = (transposed_gemm[((i_outer_inner * 8) + 2)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 22)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 6)]));
      transposed_gemm[((i_outer_inner * 8) + 3)] = (transposed_gemm[((i_outer_inner * 8) + 3)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 30)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 6)]));
      transposed_gemm[((i_outer_inner * 8) + 4)] = (transposed_gemm[((i_outer_inner * 8) + 4)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 38)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 6)]));
      transposed_gemm[((i_outer_inner * 8) + 5)] = (transposed_gemm[((i_outer_inner * 8) + 5)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 46)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 6)]));
      transposed_gemm[((i_outer_inner * 8) + 6)] = (transposed_gemm[((i_outer_inner * 8) + 6)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 54)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 6)]));
      transposed_gemm[((i_outer_inner * 8) + 7)] = (transposed_gemm[((i_outer_inner * 8) + 7)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 62)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 6)]));
      transposed_gemm[(i_outer_inner * 8)] = (transposed_gemm[(i_outer_inner * 8)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 7)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 7)]));
      transposed_gemm[((i_outer_inner * 8) + 1)] = (transposed_gemm[((i_outer_inner * 8) + 1)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 15)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 7)]));
      transposed_gemm[((i_outer_inner * 8) + 2)] = (transposed_gemm[((i_outer_inner * 8) + 2)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 23)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 7)]));
      transposed_gemm[((i_outer_inner * 8) + 3)] = (transposed_gemm[((i_outer_inner * 8) + 3)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 31)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 7)]));
      transposed_gemm[((i_outer_inner * 8) + 4)] = (transposed_gemm[((i_outer_inner * 8) + 4)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 39)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 7)]));
      transposed_gemm[((i_outer_inner * 8) + 5)] = (transposed_gemm[((i_outer_inner * 8) + 5)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 47)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 7)]));
      transposed_gemm[((i_outer_inner * 8) + 6)] = (transposed_gemm[((i_outer_inner * 8) + 6)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 55)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 7)]));
      transposed_gemm[((i_outer_inner * 8) + 7)] = (transposed_gemm[((i_outer_inner * 8) + 7)] + (A_shared[((((((int)threadIdx.x) >> 4) * 128) + (i_outer_inner * 64)) + 63)] * B_shared[(((((int)threadIdx.x) & 15) * 8) + 7)]));
    }
  }
  for (int i_inner = 0; i_inner < 16; ++i_inner) {
    out[((((((((int)blockIdx.x) >> 6) * 32768) + ((((int)threadIdx.x) >> 4) * 16384)) + (i_inner * 1024)) + ((((int)blockIdx.x) & 63) * 16)) + (((int)threadIdx.x) & 15))] = (transposed_gemm[i_inner] + C[((((((((int)blockIdx.x) >> 6) * 32768) + ((((int)threadIdx.x) >> 4) * 16384)) + (i_inner * 1024)) + ((((int)blockIdx.x) & 63) * 16)) + (((int)threadIdx.x) & 15))]);
  }
}

